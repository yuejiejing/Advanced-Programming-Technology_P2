#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime_api.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <assert.h>
#include <cstring>
#include <string>

using namespace std;

#define DEFINE_TYPE_D2 0
#define DEFINE_TYPE_D3 1
#define BLOCK_SIZE 64
#define STRING_2D string("2D")
#define STRING_3D string("3D")

// 2D
struct Struct2D
{
	int l_x;
	int l_y;
	int width;
	int height;
	float fixed_temp;
};

// 3D
struct Struct3D
{
	int l_x;
	int l_y;
	int l_z;
	int width;
	int height;
	int depth;
	float fixed_temp;
};

// Param
struct StructParam
{
	float fK;
	int iTime;
	int iWidth;
	int iHeight;
	int iDepth;
	float fStart;
	vector<Struct2D> v2D;
	vector<Struct3D> v3D;
};

// Read param
int ReadParam(const string& sFileName, StructParam& stParam)
{
	const int iSize = 1024;
	char sBuffer[iSize];
	ifstream fIn(sFileName.c_str());
	int iType;
	string sSep = ",";
	int iCount = 0;
	Struct2D st2D;
	Struct3D st3D;

	// file exist
	if( !fIn.is_open() )
	{
		return -1;
	}

	while (!fIn.eof())
	{
		fIn.getline(sBuffer, iSize);
		// comment and blank
		if (sBuffer[0] == '#' || strlen(sBuffer) == 0)
		{
			continue;
		}

		cout<<sBuffer<<endl;
		switch(iCount)
		{
			case 0:
				if(string(sBuffer) == STRING_2D)
				{
				    iType = DEFINE_TYPE_D2;
				}
				else
				{
				    iType = DEFINE_TYPE_D3;
				}
				break;
			case 1:
			    stParam.fK = atof(sBuffer);
			    break;
			case 2:
			    stParam.iTime = atoi(sBuffer);
			    break;
			case 3:
			    if( iType== DEFINE_TYPE_D2)
                {
                    sscanf(sBuffer, "%d,%d", &stParam.iWidth, &stParam.iHeight);
                }
                else
                {
                    sscanf(sBuffer, "%d,%d,%d", &stParam.iWidth, &stParam.iHeight, &stParam.iDepth);
                }
			    break;
			case 4:
			    stParam.fStart = atof(sBuffer);
			    break;
			default:
			{
				if( iType == DEFINE_TYPE_D2)
				{
					sscanf(sBuffer, "%d,%d,%d,%d,%f", &st2D.l_x, &st2D.l_y, &st2D.width, &st2D.height, &st2D.fixed_temp);
                    cout<<st2D.l_x<<" "<<st2D.l_y<<" "<<st2D.width<<" "<<st2D.height<<" "<<st2D.fixed_temp<<endl;
					stParam.v2D.push_back(st2D);
				}
				else if( iType == DEFINE_TYPE_D3 )
				{
					sscanf(sBuffer, "%d,%d,%d,%d,%d,%d,%f", &st3D.l_x, &st3D.l_y, &st3D.l_z, &st3D.width, &st3D.height, &st3D.depth, &st3D.fixed_temp);
					cout<<st3D.l_x<<" "<<st3D.l_y<<" "<<st3D.l_z<<" "<<st3D.width<<" "<<st3D.height<<" "<<st3D.depth<<" "<<st3D.fixed_temp<<endl;
					stParam.v3D.push_back(st3D);
				}
			}
				break;
		}
		iCount++;
	}
	cout<<"sFileName:"<<sFileName<<endl;
	fIn.close();
	return iType;
}

__global__ void New2Pre2D(float *pre_arr, const float *new_arr, dim3 dim)
{
	int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x + dim.x;
	pre_arr[idx] = new_arr[idx];
}

__global__ void DiffusionCUDA2D(const float *pre_arr, float *new_arr, dim3 dim, const float k)
{

	// Start from dim.x in case subscribes becomes negative
	int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x + dim.x;
	int idx_up = idx - dim.x;
	int idx_down = idx + dim.x;
	int idx_left = idx - 1;
	int idx_right = idx + 1;

	bool left_edge = idx % dim.x == 0;
	bool right_edge = (idx + 1) % dim.x == 0;
	bool top_edge = idx < 2 * dim.x;
	bool bottom_edge = (idx >= dim.x * dim.y) && (idx < dim.x * (dim.y + 1));

	if (idx < dim.x * (dim.y + 1))
	{
		new_arr[idx] = pre_arr[idx] + k * (pre_arr[idx_up] * (!top_edge) + pre_arr[idx_down] * (!bottom_edge) +
			pre_arr[idx_left] * (!left_edge) + pre_arr[idx_right] * (!right_edge) - 4 * pre_arr[idx] +
			pre_arr[idx] * left_edge + pre_arr[idx] * right_edge + pre_arr[idx] * top_edge + pre_arr[idx] * bottom_edge);
	}
}

__global__ void RecoverFixed2D(float *new_arr, dim3 dim, int x, int y, int width, int height, float temp)
{
	int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int offset_x = idx % width;
	int offset_y = idx / width;

	if (idx < width * height)
	{
		new_arr[(y + offset_y) * dim.x + x + offset_x] = temp;
	}
}


__global__ void New2Pre3D(float *pre_arr, const float *new_arr, dim3 dim) {
	int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x + dim.x * dim.y;
	pre_arr[idx] = new_arr[idx];
}


__global__ void DiffusionCUDA3D(const float *pre_arr, float *new_arr, dim3 dim, const float k)
{

	// Start from dim.x in case subscribes becomes negative
	int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x + dim.x * dim.y;
	int idx_up = idx - dim.x;
	int idx_down = idx + dim.x;
	int idx_left = idx - 1;
	int idx_right = idx + 1;
	int idx_inner = idx - dim.x * dim.y;
	int idx_outer = idx + dim.x * dim.y;

	bool left_plane = idx % dim.x == 0;
	bool right_plane = (idx + 1) % dim.x == 0;
	bool upper_plane = idx % (dim.x * dim.y) < dim.x;
	bool bottom_plane = idx % (dim.x * dim.y) >= dim.x * (dim.y - 1);
	bool inner_plane = idx < 2 * dim.x * dim.y;
	bool outer_plane = (idx >= dim.x * dim.y * dim.z) && (idx < dim.x * dim.y * (dim.z + 1));

	if (idx < dim.x * dim.y * (dim.z + 1))
	{
		new_arr[idx] = pre_arr[idx] + k * (pre_arr[idx_up] * (!upper_plane) + pre_arr[idx_down] * (!bottom_plane) +
			pre_arr[idx_left] * (!left_plane) + pre_arr[idx_right] * (!right_plane) +
			pre_arr[idx_inner] * (!inner_plane) + pre_arr[idx_outer] * (!outer_plane) - 
			6 * pre_arr[idx] + pre_arr[idx] * left_plane + pre_arr[idx] * right_plane + 
			pre_arr[idx] * upper_plane + pre_arr[idx] * bottom_plane + pre_arr[idx] * inner_plane + 
			pre_arr[idx] * outer_plane);
	}
}


__global__ void RecoverFixed3D(float *new_arr, dim3 dim, int x, int y, int z, int width, int height, int depth, float temp)
{
	int idx = blockIdx.x * BLOCK_SIZE + threadIdx.x;
	int offset_x = (idx % (height * width)) % width;
	int offset_y = (idx % (height * width)) / width;
	int offset_z = idx / (height * width);

	if (idx < width * height * depth)
	{
		new_arr[(z + offset_z) * dim.x * dim.y + (y + offset_y) * dim.x + x + offset_x] = temp;
	}
}


Struct2D * parseVec(const vector<Struct2D> &vec)
{
	Struct2D * p = (Struct2D*)malloc(sizeof(Struct2D) * vec.size());
	for (int i = 0; i < vec.size(); ++i)
	{
		p[i] = vec[i];
	}
	return p;
}

Struct3D * parseVec(const vector<Struct3D> &vec)
{
	Struct3D * p = (Struct3D*)malloc(sizeof(Struct3D) * vec.size());
	for (int i = 0; i < vec.size(); ++i)
	{
		p[i] = vec[i];
	}
	return p;
}

void usage(const string& sBin)
{
	cout<<sBin<<" 2d.conf"<<endl;
}

// Deal 2D
void Deal2D(StructParam& stParam, const string& sFile)
{
	Struct2D *tvec = parseVec(stParam.v2D);

	// graph
	dim3 dim(stParam.iWidth, stParam.iHeight, 1);
	unsigned int iLength = dim.x * dim.y;
	unsigned int iMemSize = sizeof(float) * (iLength + dim.x);

	float *pre_graph = reinterpret_cast<float *>(malloc(iMemSize));
	float *new_graph = reinterpret_cast<float *>(malloc(iMemSize));
	for (int i = 0; i < dim.y + 1; ++i)
	{
		for (int j = 0; j < dim.x; ++j)
		{
			pre_graph[i * dim.x + j] = stParam.fStart;
		}
	}

	for(int k = 0; k < stParam.v2D.size(); k++)
	{
		Struct2D& st2D = stParam.v2D[k];
		for(int i = st2D.l_y + 1; i < st2D.l_y + 1 + st2D.height; i++)
		{
			for(int j = st2D.l_x; j < st2D.l_x + st2D.width; j++)
			{
				pre_graph[i * dim.x + j] = st2D.fixed_temp;
			}
		}
	}

	cout<<11<<endl;

	int iVecSize = stParam.v2D.size();
	float *d_pre, *d_new;

	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_pre), iMemSize));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_new), iMemSize));
	checkCudaErrors(hipMemcpy(d_pre, pre_graph, iMemSize, hipMemcpyHostToDevice));

	for (int t = 0; t < stParam.iTime; ++t)
	{
		DiffusionCUDA2D <<< (iLength + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (d_pre, d_new, dim, stParam.fK);
		for (int i = 0; i < iVecSize; ++i)
		{
			RecoverFixed2D <<< (stParam.v2D[i].width * stParam.v2D[i].height + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (d_new, dim,
					stParam.v2D[i].l_x, stParam.v2D[i].l_y + 1, stParam.v2D[i].width, stParam.v2D[i].height, stParam.v2D[i].fixed_temp);
		}
		New2Pre2D <<< (iLength + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (d_pre, d_new, dim);
	}
	checkCudaErrors(hipMemcpy(new_graph, d_new, iMemSize, hipMemcpyDeviceToHost));

	cout<<111<<endl;
	// Write file
	ofstream fOut;
	fOut.open(sFile.c_str(), ios::out);
	assert(fOut);
	for (int i = 1; i < stParam.iHeight + 1; ++i)
	{
		for (int j = 0; j < stParam.iWidth - 1; ++j)
		{
			fOut << new_graph[i * stParam.iWidth + j] << ", ";
		}
		if (i != stParam.iHeight)
		{
			fOut << new_graph[i * stParam.iWidth + stParam.iWidth - 1] << endl;
		}
		else
		{
			fOut << new_graph[i * stParam.iWidth + stParam.iWidth - 1];
		}
	}

	// clean
	fOut.close();
	free(new_graph);
	free(pre_graph);
	checkCudaErrors(hipFree(d_pre));
	checkCudaErrors(hipFree(d_new));
}



// Function to manipulate 3D heat diffusion
void Deal3D(StructParam& stParam, const string& sFile)
{
	Struct3D * tvec = parseVec(stParam.v3D);
	dim3 dim(stParam.iWidth, stParam.iHeight, stParam.iDepth);
	unsigned int iLength = dim.x * dim.y * dim.z;
	unsigned int iMemSize = sizeof(float) * (iLength + dim.x * dim.y);

	float *pre_graph = reinterpret_cast<float *>(malloc(iMemSize));
	float *new_graph = reinterpret_cast<float *>(malloc(iMemSize));

	for (int m = 0; m < stParam.iDepth; ++m)
	{
		for (int i = 0; i < stParam.iHeight; ++i)
		{
			for (int j = 0; j < stParam.iWidth; ++j)
			{
				pre_graph[m * stParam.iHeight * stParam.iWidth + i * stParam.iWidth + j] = stParam.fStart;
			}
		}
	}

	for(int k = 0; k < stParam.v3D.size(); k++)
	{
		Struct3D& st3D = stParam.v3D[k];
		for (int m = st3D.l_z + 1; m < st3D.l_z + st3D.depth + 1; ++m)
		{
			for (int i = st3D.l_y; i < st3D.l_y + st3D.height; ++i)
			{
				for (int j = st3D.l_x; j < st3D.l_x + st3D.width; ++j)
				{
					pre_graph[m * stParam.iHeight * stParam.iWidth + i * stParam.iWidth + j] = st3D.fixed_temp;
				}
			}
		}
	}
	
	int iVecSize = stParam.v3D.size();
	float *d_pre, *d_new;

	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_pre), iMemSize));
	checkCudaErrors(hipMalloc(reinterpret_cast<void **>(&d_new), iMemSize));
	checkCudaErrors(hipMemcpy(d_pre, pre_graph, iMemSize, hipMemcpyHostToDevice));
	for (int t = 0; t < stParam.iTime; ++t)
	{
		DiffusionCUDA3D <<< (iLength + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (d_pre, d_new, dim, stParam.fK);
		for (int i = 0; i < iVecSize; ++i)
		{
			RecoverFixed3D <<< (stParam.v3D[i].width * stParam.v3D[i].height * stParam.v3D[i].depth + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>>(
				d_new, dim, stParam.v3D[i].l_x, stParam.v3D[i].l_y, stParam.v3D[i].l_z + 1, stParam.v3D[i].width, stParam.v3D[i].height,
						stParam.v3D[i].depth, stParam.v3D[i].fixed_temp);
		}
		New2Pre3D <<< (iLength + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >>> (d_pre, d_new, dim);
	}
	checkCudaErrors(hipMemcpy(new_graph, d_new, iMemSize, hipMemcpyDeviceToHost));

	// Write file
	ofstream fOut;
	fOut.open(sFile.c_str(), ios::out);
	for (int m = 1; m < stParam.iDepth + 1; ++m)
	{
		for (int i = 0; i < stParam.iHeight; ++i)
		{
			for (int j = 0; j < stParam.iWidth - 1; ++j)
			{
				fOut << new_graph[m * stParam.iHeight * stParam.iWidth + i * stParam.iWidth + j] << ", ";
			}
			if (m == stParam.iDepth && i == stParam.iHeight - 1)
			{
				fOut << new_graph[m * stParam.iHeight * stParam.iWidth + i * stParam.iWidth + stParam.iWidth - 1];
			}
			else
			{
				fOut << new_graph[m * stParam.iHeight * stParam.iWidth + i * stParam.iWidth + stParam.iWidth - 1] << endl;
			}
		}
		if (m != stParam.iDepth)
		{
			fOut << endl;
		}
	}

	fOut.close();
	free(new_graph);
	free(pre_graph);
	checkCudaErrors(hipFree(d_pre));
	checkCudaErrors(hipFree(d_new));
}

int main(int argc, char** argv)
{
	if( argc != 2 )
	{
		usage(argv[0]);
		return -1;
	}

	// Read Parameters
	StructParam stParam;
	string sFileName = argv[1];
	string sOutFile = "out" + sFileName + ".csv";
	int iType = ReadParam(sFileName, stParam);

	cout<<stParam.iTime<<" "<<stParam.fK<<" "<<stParam.iWidth<<" "<<stParam.iHeight<<" "<<stParam.iDepth<<" "<<stParam.fStart<<" "<<stParam.v2D.size()<<" "<<stParam.v3D.size()<<endl;
	switch(iType)
	{
		case DEFINE_TYPE_D2:
			Deal2D(stParam, sOutFile);
			break;
		case DEFINE_TYPE_D3:
			Deal3D(stParam, sOutFile);
			break;
		default:
			cout<<"iType:"<<iType<<endl;
			break;
	}
	return 0;
}